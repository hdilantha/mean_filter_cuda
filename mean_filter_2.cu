
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>



void mean_filter_h(int **img, int **res, int N, int M, int k)
{
  int count;
  float temp;
  for(int n = 0; n < N; n++) {
    for(int m = 0; m < M; m++) {
      count = 0;
      temp = 0.0;
      for(int i = N - k; i <= N + k; i++) {
        for(int j = M - k; j <= M + k; j++) {
          if(i >= 0 && i < N && j >= 0 && j < M) {
              count = count + 1;
              temp = res[n][m] + img[i][j];
          }
        }
        temp = temp / count;
        res[n][m] = (int)temp;
      }
    }
  }
}

int main()
{
  const int N = 4;
  const int M = 4;
  int k = 3;

  int *a[N], *b[N];

  for(int i = 0; i < N; i++ ) {
    a[i] = (int *)malloc(M * sizeof(int));
    b[i] = (int *)malloc(M * sizeof(int));
  }

  for(int i = 0; i < N; i++ ) {
    for(int j = 0; j < M; j++ ) {
      a[i][j] = 1;
      b[i][j] = 0;
    }
  }


  //clock_t start_h = clock();
  mean_filter_h(a, b, N, M, k / 2);
  //clock_t end_h = clock();

  //double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;
  //printf("CPU Time: %f\n", time_h);

  for(int i = 0; i < N; i++ ) {
    for(int j = 0; j < M; j++ ) {
      printf("%d ", b[i][j]);
    }
  }

  return 0;
}
