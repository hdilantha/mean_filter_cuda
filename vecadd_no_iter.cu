#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

int *a, *b;  // host data
int *c, *c2;  // results

__global__ void vecAdd(int *A, int *B, int *C, int N)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  C[tid] = A[tid] + B[tid];
}

void vecAdd_h(int *A1, int *B1, int *C1, int N)
{
  for(int i = 0; i < N; i++)
    C1[i] = A1[i] * B1[i];
}

int main(int argc, char **argv)
{
  // printf("Begin \n");
  int v_sizes [4] = {100, 10000, 1000000, 10000000};

  for(int j = 0; j < 4; j++) {
    int n = v_sizes[j];
    int nBytes = n * sizeof(int);
    int block_size, block_no;
    a = (int *)malloc(nBytes);
    b = (int *)malloc(nBytes);
    c = (int *)malloc(nBytes);
    c2 = (int *)malloc(nBytes);
    int *a_d, *b_d, *c_d;
    block_size = 100;
    block_no = n / block_size;
    dim3 dimBlock(block_size, 1, 1);
    dim3 dimGrid(block_no, 1, 1);

    for(int i = 0; i < n; i++ ) {
      a[i] = sin(i) * sin(i);
      b[i] = cos(i) * cos(i);
    }

    // printf("Allocating device memory on device..\n");
    hipMalloc((void **)&a_d, n * sizeof(int));
    hipMalloc((void **)&b_d, n * sizeof(int));
    hipMalloc((void **)&c_d, n * sizeof(int));

    // printf("Copying to device..\n");
    hipMemcpy(a_d, a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, nBytes, hipMemcpyHostToDevice);

    // printf("Doing GPU Vector add..\n");
    clock_t start_d = clock();
    vecAdd<<<block_no, block_size>>>(a_d, b_d, c_d, n);
    hipDeviceSynchronize();
    clock_t end_d = clock();

    // printf("Doing CPU Vector add..\n");
    clock_t start_h = clock();
    vecAdd_h(a, b, c2, n);
    clock_t end_h = clock();

    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;
    hipMemcpy(c, c_d, nBytes, hipMemcpyDeviceToHost);

    printf("Number of elements: %d, GPU Time: %f, CPU Time: %f\n", n, time_d, time_h);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
  }

  return 0;
}
