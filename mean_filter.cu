#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

// int *a, *b;  // host data


__global__ void mean_filter(int *A, int *B, int *C, int N)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  C[tid] = A[tid] + B[tid];
}

void mean_filter_h(int *img, int *res, int N, int M, int k)
{
  for(int n = 0; n < N; n++) {
    for(int m = 0; m < M; m++) {
      for(int i = N - k; i <= N + k; i++) {
        for(int j = M - k; j <= M + k; j++) {
          if(img[i][j]) {
              res[n][m] = res[n][m] + img[i][j];
          }
        }
      }
    }
  }
}

int main(int argc, char **argv)
{
  // printf("Begin \n");
  const int N = 4;
  const int M = 4;
  int k = 3;

  int a[N][M] = {
    {0, 1, 2, 3},
    {4, 5, 6, 7},
    {8, 9, 10, 11},
    {12, 13, 14, 15}
  };

  int b[N][M] = {
    {0, 0, 0, 0},
    {0, 0, 0, 0},
    {0, 0, 0, 0},
    {0, 0, 0, 0}
  };
  // b = (int *)malloc(nBytes);

  // int *a_d, *b_d, *c_d;
  // block_size = 100;
  // block_no = n / block_size;

  // printf("Allocating device memory on device..\n");
  // hipMalloc((void **)&a_d, n * sizeof(int));
  // hipMalloc((void **)&b_d, n * sizeof(int));
  // hipMalloc((void **)&c_d, n * sizeof(int));

  // printf("Copying to device..\n");
  // hipMemcpy(a_d, a, nBytes, hipMemcpyHostToDevice);
  // hipMemcpy(b_d, b, nBytes, hipMemcpyHostToDevice);

  // printf("Doing GPU Vector add..\n");
  // clock_t start_d = clock();
  // vecAdd<<<block_no, block_size>>>(a_d, b_d, c_d, n);
  // hipDeviceSynchronize();
  // clock_t end_d = clock();

  // printf("Doing CPU Vector add..\n");
  clock_t start_h = clock();
  mean_filter_h((int *)a, (int *)b, N, M, k / 2);
  clock_t end_h = clock();

  // double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
  double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;
  // hipMemcpy(c, c_d, nBytes, hipMemcpyDeviceToHost);

  printf("CPU Time: %f\n", time_h);
  // printf("Number of elements: %d, GPU Time: %f, CPU Time: %f\n", n, time_d, time_h);
  // hipFree(a_d);
  // hipFree(b_d);
  // hipFree(c_d);

  for(int i = 0; i < N; i++ ) {
    for(int j = 0; j < M; j++ ) {
      printf("CPU Time: %f\n", b[i][j]);
    }
  }

  return 0;
}
